#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576
int data[DATA_SIZE];

void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++)
	{
		number[i] = rand() % 10;
	}
}

__global__ static void sumOfSquares(int *num, int* result)
{
	int sum = 0;
	int i;

	for (i = 0; i < DATA_SIZE; i++)
	{
		sum += num[i] * num[i];
	}
	*result = sum;

}

// output given hipDeviceProp_t
void OutputSpec(const hipDeviceProp_t sDevProp)
{
	printf("1. �binitCUDA��Ū�X\n");
	printf("1). Device name: %s\n", sDevProp.name);
	printf("2). Total amount of global memory: %.0f MBytes (%llu bytes)\n",
		(float)sDevProp.totalGlobalMem / DATA_SIZE, (unsigned long long) sDevProp.totalGlobalMem);
	printf("3). Maximum number of threads per multiprocessor:  %d\n", sDevProp.maxThreadsPerMultiProcessor);
	printf("3). Maximum number of threads per block:           %d\n", sDevProp.maxThreadsPerBlock);
	printf("3). Max dimension size of a thread block (x,y,z): ( %d, %d, %d )\n", sDevProp.maxThreadsDim[0], sDevProp.maxThreadsDim[1], sDevProp.maxThreadsDim[2]);
	printf("4). GPU Max Clock rate: %.0f MHz (%0.2f GHz)\n\n", sDevProp.clockRate * 1e-3f, sDevProp.clockRate * 1e-6f);
}


void main()
{
	// part1, check the number of device
	int  iDeviceCount = 0;
	hipGetDeviceCount(&iDeviceCount);
	//printf("Number of GPU: %d\n", iDeviceCount);

	if (iDeviceCount == 0)
	{
		printf("No supported GPU\n");
		return;
	}

	// part2, output information of each device
	for (int i = 0; i < iDeviceCount; ++i)
	{
		//printf("\n=== Device %i ===\n", i);
		hipDeviceProp_t  sDeviceProp;
		hipGetDeviceProperties(&sDeviceProp, i);
		OutputSpec(sDeviceProp);
	}


	//run of GPU
	GenerateNumbers(data, DATA_SIZE);
	int* gpudata, *result, sum;

	hipEvent_t go, stop;
	float time_gpu;
	hipEventCreate(&go);
	hipEventCreate(&stop);

	hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));

	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

	hipEventRecord(go, 0);
	sumOfSquares << <1, 1, 0 >> >(gpudata, result);
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);

	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(data, gpudata, sizeof(float)*DATA_SIZE, hipMemcpyDeviceToHost);

	hipFree(gpudata);
	hipFree(result);
	hipEventElapsedTime(&time_gpu, go, stop);
	time_gpu = time_gpu / 1000.0;

	printf("2. �bGPU�MCPU�W���O�p�⵲�G\n");
	printf("sum (GPU): %d\n", sum);
	//run of CPU
	int cpu_time = 0;
	sum = 0;
	unsigned long start = clock();
	for (int i = 0; i < DATA_SIZE; i++)
	{
		sum += data[i] * data[i];
	}
	unsigned long end = clock();
	
	printf("sum (CPU): %d\n\n", sum);
	printf("3. GPU�MCPU�p�����ɶ�\n");
	printf("time (GPU) %1.3fs\n",time_gpu);
	printf("time (CPU): %1.3fs\n\n", (end - start) / 1000.0);

}